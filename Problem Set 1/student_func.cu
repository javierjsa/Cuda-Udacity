#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this homework.

//Each channel Red, Blue, Green, and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>
#include <iostream>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  //blockDim.x*blockIdx.x + threadIdx.x


  int block_start_row = blockIdx.y * numCols * blockDim.y;
  int block_start_col = blockIdx.x * blockDim.x;
 
  int thread_row = block_start_row + threadIdx.y * numCols;
  int thread_col = block_start_col + threadIdx.x;
  
  int thread_pos = thread_row+thread_col;
  
  if (thread_pos<numRows*numCols){
  
     uchar4 rgba = rgbaImage[thread_pos];
     float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
     greyImage[thread_pos] = channelSum;  
  }     

}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  
  const int block_size = 32;
  
  int block_rows, block_cols;

  if (numRows % block_size >0) 
     block_rows = (numRows/block_size)+ 1;
  else
     block_rows = (numRows/block_size);

  if (numCols % block_size >0) 
     block_cols = int(numCols/block_size)+ 1;
  else
     block_cols = int(numCols/32);
 
  //std::cout<<"Rows: "<<block_rows<<" Cols: "<< block_cols<<"\n";
  //std::cout<<"numRows: "<<numRows<<" numCols: "<< numCols;
  
  //Allocate memory
  const dim3 blockSize(block_size,block_size, 1);  //TODO
  const dim3 gridSize( block_cols, block_rows, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

